#include "hip/hip_runtime.h"
// CUDA
#include "hip/hip_runtime.h"
#include ""

// Helpers
#include "hip/hip_runtime_api.h"


// Work around false error squiggly lines inside VS; see: https://stackoverflow.com/a/27992604
#ifdef __INTELLISENSE__
#define KERNEL_2ARGS(grid, block)
#define KERNEL_3ARGS(grid, block, sh_mem)
#define KERNEL_4ARGS(grid, block, sh_mem, stream)
#else
#define KERNEL_2ARGS(grid, block) <<< grid, block >>>
#define KERNEL_3ARGS(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_4ARGS(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#endif


/**
 * Get an arbitrary 3D color vec from a 2D numeric input.
 */
__device__ uchar3 toColor(float2 texCoords, int imageW, int imageH)
{
	return uchar3{
		int(255.99f * texCoords.x / float(imageW)),
		int(255.99f * texCoords.y / float(imageH)),
		int(255.99f * .3f)
	};
}


__global__ void jumpFloodKernel(uchar3* pixelCanvas, float2* numericCanvas, int diagramXDim, int diagramYDim)
{
	// calculate non-normalized texture coordinates
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

	// Ignore out-of-bounds index
	if (x > diagramXDim || y > diagramYDim) return;

	float maximalDim = fmaxf(diagramXDim, diagramYDim);

	// JFA pass(es) loop
	for (int passIndex = 0; passIndex < log2f(maximalDim); ++passIndex)
	{
		float step = powf(2.f, (log2f(maximalDim) - passIndex - 1.f));

		// At first, the best candidate is ourselves
		unsigned selfIdx = y * diagramXDim + x;
		float2 closestCandidate = numericCanvas[selfIdx];
		float closestDistance = float(INT_MAX);

		// JFA pass computations
		for (int gridY = 0; gridY < 3; ++gridY)
		{
			for (int gridX = 0; gridX < 3; ++gridX)
			{
				float xLookup = x - step + gridX * step;
				float yLookup = y - step + gridY * step;

				// Ignore out-of-bounds
				if (xLookup < 1e-6f || xLookup > diagramXDim || yLookup < 1e-6f || yLookup > diagramYDim) continue;

				int lookupIdx = yLookup * diagramXDim + xLookup;
				float2 otherCandidate = numericCanvas[lookupIdx];

				if (otherCandidate.x + otherCandidate.y > 1e-6f)
				{
					float otherDistance = sqrtf(
						(otherCandidate.x - x) * (otherCandidate.x - x)
						 + (otherCandidate.y - y) * (otherCandidate.y - y)
					);
					if (otherDistance < closestDistance)
					{
						closestCandidate = otherCandidate;
						closestDistance = otherDistance;
					}
				}

				// Abandoned idea using texture objects...
				#if 0
				#ifndef __INTELLISENSE__
								rTheirs = tex2D<float>(texture, 2 * xLookup, yLookup);
								gTheirs = tex2D<float>(texture, 2 * xLookup + 1, yLookup);
				#endif
				#endif
				#if 0
				#ifndef __INTELLISENSE__
								rMe = tex2D<float>(texture, 2 * x, y);
								gMe = tex2D<float>(texture, 2 * x + 1, y);
				#endif
				#endif
			}
		}

		pixelCanvas[selfIdx] = toColor(closestCandidate, diagramXDim, diagramYDim);
		numericCanvas[selfIdx] = closestCandidate;

#ifndef __INTELLISENSE__
		__syncthreads();
#endif
	}
}

void jumpFloodWithCuda(unsigned char* hostPixelCanvas, float2* hostNumericCanvas, int diagramXDim, int diagramYDim)
{
	// For sanity
	checkCudaErrors(hipSetDevice(0));

	int pixelChannels = 3;
	int numericChannels = 2;

	// Allocate device numeric canvas
	float2* deviceNumericCanvas;
	size_t numericCanvasSize = diagramXDim * diagramYDim * numericChannels * sizeof(float);
	checkCudaErrors(hipMalloc((void**)&deviceNumericCanvas, numericCanvasSize));
	checkCudaErrors(hipMemset(deviceNumericCanvas, 0, numericCanvasSize));


	// Allocate device pixel canvas
	uchar3* devicePixelCanvas;
	size_t pixelCanvasSize = diagramXDim * diagramYDim * pixelChannels * sizeof(unsigned char);
	checkCudaErrors(hipMalloc((void**)&devicePixelCanvas, pixelCanvasSize));
	checkCudaErrors(hipMemset(devicePixelCanvas, 0, pixelCanvasSize));

#if 0
	// Allocate image space; see: https://stackoverflow.com/a/16217548
	hipArray* deviceInputCanvas;
	hipChannelFormatDesc channelDescription = hipCreateChannelDesc<float>();
	checkCudaErrors(hipMallocArray(&deviceInputCanvas, &channelDescription, diagramXDim * numericChannels, diagramYDim));
    checkCudaErrors(hipMemcpy2DToArray(
        deviceInputCanvas,                                             // Dest data hipArray
        0,
        0,
        hostNumericCanvas,                                             // Source data pointer
        diagramXDim * numericChannels * sizeof(float),                 // Pitch/alignment for this allocated memory
        diagramXDim * numericChannels * sizeof(float),                 // Copy span width (bytes)
        diagramYDim,                                                   // Copy span height (elements)
        hipMemcpyHostToDevice
    ));

	// Texture resource
	hipResourceDesc textureResource;
	memset(&textureResource, 0, sizeof(hipResourceDesc));
	textureResource.resType         = hipResourceTypeArray;
	textureResource.res.array.array = deviceInputCanvas;

	// Texture description (actually important); see: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-object-api
	hipTextureDesc textureDescription;
	memset(&textureDescription, 0, sizeof(hipTextureDesc));
	textureDescription.addressMode[0]   = hipAddressModeClamp;         // Clamp over-index lookups across X (default)
	textureDescription.addressMode[1]   = hipAddressModeClamp;         // Clamp over-index lookups across Y (default)
    textureDescription.filterMode       = hipFilterModePoint;          // Do not interpolate, pick the actual closest value during lookup
	textureDescription.readMode         = hipReadModeElementType;      // Do not convert the returned lookup data
	textureDescription.normalizedCoords = 0;                            // Use tex coords, not coords between [0,1)

	// Texture object (allocated and bound during runtime, as opposed to the texture reference)
	hipTextureObject_t texture = 0;
	checkCudaErrors(hipCreateTextureObject(&texture, &textureResource, &textureDescription, NULL));
#endif

	// Copy into
	checkCudaErrors(hipMemcpy2D(
		deviceNumericCanvas,
		diagramXDim * numericChannels * sizeof(float),
		hostNumericCanvas,
		diagramXDim * numericChannels * sizeof(float),
		diagramXDim * numericChannels * sizeof(float),
		diagramYDim,
		hipMemcpyHostToDevice
	));


	// Define dimensions & launch kernel
	dim3 bDim(32, 32, 1);
	dim3 gDim(diagramXDim / bDim.x, diagramYDim / bDim.y, 1);
	jumpFloodKernel KERNEL_2ARGS(gDim, bDim)(devicePixelCanvas, deviceNumericCanvas, diagramXDim, diagramYDim);

	// Sanity checks
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("Kernel launch failed!");

	// Copy back
	checkCudaErrors(hipMemcpy2D(
		hostPixelCanvas, // Dest data pointer
		diagramXDim * pixelChannels * sizeof(unsigned char), // Dest mem alignment
		devicePixelCanvas, // Source data pointer
		diagramXDim * pixelChannels * sizeof(unsigned char), // Source mem alignment
		diagramXDim * pixelChannels * sizeof(unsigned char), // Copy span width (bytes)
		diagramYDim, // Copy span height (elements)
		hipMemcpyDeviceToHost
	));

	// Cleanup
	checkCudaErrors(hipFree(devicePixelCanvas));
	checkCudaErrors(hipFree(deviceNumericCanvas));
	//checkCudaErrors(hipFreeArray(deviceInputCanvas));
	//checkCudaErrors(hipDestroyTextureObject(texture));
	checkCudaErrors(hipDeviceReset());
}
